#include "hip/hip_runtime.h"
#include "Computations.cuh"
glm::vec3 *d_HelperVelocitiesPrey;
glm::vec3 *d_HelperVelocitiesPredators;
SimulationValues *d_values;

void Compute::init(int preyNumber, int predatorsNumber)
{
    hipMalloc((void **)&d_HelperVelocitiesPrey, preyNumber * sizeof(glm::vec3));
    hipMalloc((void **)&d_HelperVelocitiesPredators, predatorsNumber * sizeof(glm::vec3));
    hipMalloc((void **)&d_values, sizeof(SimulationValues));
}
void Compute::free()
{
    hipFree(d_HelperVelocitiesPrey);
    hipFree(d_HelperVelocitiesPredators);
    hipFree(d_values);
}

// Applies the principles of flocking behavior to a given shoal,
// defined as set of positions and vectors.
__device__ void ComputeFlockingBehaviorVelocities(glm::vec3 *positions, glm::vec3 *velocities, glm::vec3 *helperVelocities,
                                                  SpecificShoalValues *values, int id, float dt)
{
    glm::vec3 averageVelocity = glm::vec3(0, 0, 0);
    glm::vec3 averagePosition = glm::vec3(0, 0, 0);
    glm::vec3 smallAveragePosition = glm::vec3(0, 0, 0);
    int count = 0;
    int smallCount = 0;
    for (int i = 0; i < values->Number; i++)
    {
        float distance = glm::distance(positions[id], positions[i]);
        if (id != i && distance < values->Perception.BigPerceptionRadius)
        {
            averageVelocity += velocities[i];
            averagePosition += positions[i];
            count++;
            if (distance < values->Perception.SmallPerceptionRadius)
            {
                smallAveragePosition += positions[i];
                smallCount++;
            }
        }
    }

    helperVelocities[id] = velocities[id];
    if (count > 0)
    {
        averageVelocity /= count;
        averagePosition /= count;
        glm::vec3 SteeringAlignment = averageVelocity - velocities[id];
        glm::vec3 SteeringCohesion = averagePosition - positions[id];
        SteeringAlignment = glm::normalize(SteeringAlignment);
        SteeringAlignment *= values->Forces.AlignmentForce;
        SteeringCohesion = glm::normalize(SteeringCohesion);
        SteeringCohesion *= values->Forces.CohesionForce;
        helperVelocities[id] += SteeringAlignment * dt + SteeringCohesion * dt;
    }
    if (smallCount > 0)
    {
        smallAveragePosition /= smallCount;
        glm::vec3 SteeringSeparation = positions[id] - smallAveragePosition;
        SteeringSeparation = glm::normalize(SteeringSeparation);
        SteeringSeparation *= values->Forces.SeparationForce;
        helperVelocities[id] += SteeringSeparation * dt;
    }

    helperVelocities[id] = glm::normalize(helperVelocities[id]);
    helperVelocities[id] *= values->Speed;
}

// Used to make smaller type of fish follow worm when it's visible.
__device__ void ComputePreyHungerVelocities(glm::vec3 *positions, glm::vec3 *helperVelocities,
                                            SimulationValues *values, int id, float dt)
{
    if (values->Settings.CursorVisible)
    {
        if (glm::distance(positions[id], values->Environment.CursorPosition) <= values->Prey.Perception.FoodPerceptionRadius)
        {
            glm::vec3 SteeringHunger = values->Environment.CursorPosition - positions[id];
            SteeringHunger = glm::normalize(SteeringHunger);
            SteeringHunger *= values->Prey.Forces.HungerForce;
            helperVelocities[id] += SteeringHunger * dt;
        }
    }
}

// Used to make smaller fish escape from bigger fish when in proximity.
__device__ void ComputePreyFearVelocities(glm::vec3 *positions, glm::vec3 *helperVelocities,
                                          SimulationValues *values, int id, float dt, glm::vec3 *predatorPositions)
{
    glm::vec3 averagePosition = glm::vec3(0, 0, 0);
    int count = 0;
    for (int i = 0; i < values->Predators.Number; i++)
    {
        float distance = glm::distance(positions[id], predatorPositions[i]);
        {
            if (distance < values->Prey.Perception.DangerPerceptionRadius)
            {
                averagePosition += predatorPositions[i];
                count++;
            }
        }
    }
    if (count > 0)
    {
        averagePosition /= count;
        glm::vec3 SteeringFear = positions[id] - averagePosition;
        SteeringFear = glm::normalize(SteeringFear);
        SteeringFear *= values->Prey.Forces.FearForce;
        helperVelocities[id] += SteeringFear * dt;
    }
}

// Used to make bigger fish chase smaller fish when in proximity.
__device__ void ComputePredatorHungerVelocities(glm::vec3 *positions, glm::vec3 *helperVelocities,
                                                SimulationValues *values, int id, float dt, glm::vec3 *preyPositions)
{
    glm::vec3 averagePosition = glm::vec3(0, 0, 0);
    int count = 0;
    for (int i = 0; i < values->Prey.Number; i++)
    {
        float distance = glm::distance(positions[id], preyPositions[i]);
        {
            if (distance < values->Predators.Perception.FoodPerceptionRadius)
            {
                averagePosition += preyPositions[i];
                count++;
            }
        }
    }
    if (count > 0)
    {
        averagePosition /= count;
        glm::vec3 SteeringHunger = averagePosition - positions[id];
        SteeringHunger = glm::normalize(SteeringHunger);
        SteeringHunger *= values->Predators.Forces.HungerForce;
        helperVelocities[id] += SteeringHunger * dt;
    }
}

// Applies all types of behaviors to smaller fish.
__global__ void ComputePreyVelocities(glm::vec3 *positions, glm::vec3 *velocities, glm::vec3 *helperVelocities,
                                      float dt, SimulationValues *values, glm::vec3 *predatorPositions)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < values->Prey.Number)
    {
        ComputeFlockingBehaviorVelocities(positions, velocities, helperVelocities, &values->Prey, id, dt);
        ComputePreyHungerVelocities(positions, helperVelocities, values, id, dt);
        ComputePreyFearVelocities(positions, helperVelocities, values, id, dt, predatorPositions);
    }
}

// Applies all types of behaviors to bigger fish.
__global__ void ComputePredatorsVelocities(glm::vec3 *positions, glm::vec3 *velocities, glm::vec3 *helperVelocities,
                                           float dt, SimulationValues *values, glm::vec3 *preyPositions)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < values->Predators.Number)
    {
        ComputeFlockingBehaviorVelocities(positions, velocities, helperVelocities, &values->Predators, id, dt);
        ComputePredatorHungerVelocities(positions, helperVelocities, values, id, dt, preyPositions);
    }
}

// After calculating the velocities this kernel copies them from temporary vector and adds the
// to current position. Next, container bounds are checked.
__global__ void SetPositions(glm::vec3 *positions, glm::vec3 *velocities, glm::vec3 *helperVelocities,
                             SpecificShoalValues *shoalValues, float dt, SimulationValues *values)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < shoalValues->Number)
    {
        helperVelocities[id] = glm::normalize(helperVelocities[id]);
        helperVelocities[id] *= shoalValues->Speed;
        velocities[id] = helperVelocities[id];
        positions[id] += helperVelocities[id] * dt;

        if (values->Settings.SphericalContainer)
        {
            if (glm::length(positions[id]) > values->Environment.SizeOfContainer / 2)
            {
                positions[id] = glm::normalize(positions[id]);
                positions[id] *= -values->Environment.SizeOfContainer / 2;
            }
        }
        else
        {
            if (positions[id].x < -values->Environment.SizeOfContainer / 2)
                positions[id].x = values->Environment.SizeOfContainer / 2;
            if (positions[id].x > values->Environment.SizeOfContainer / 2)
                positions[id].x = -values->Environment.SizeOfContainer / 2;
            if (positions[id].y < -values->Environment.SizeOfContainer / 2)
                positions[id].y = values->Environment.SizeOfContainer / 2;
            if (positions[id].y > values->Environment.SizeOfContainer / 2)
                positions[id].y = -values->Environment.SizeOfContainer / 2;
            if (positions[id].z < -values->Environment.SizeOfContainer / 4)
                positions[id].z = values->Environment.SizeOfContainer / 4;
            if (positions[id].z > values->Environment.SizeOfContainer / 4)
                positions[id].z = -values->Environment.SizeOfContainer / 4;
        }
    }
}

// All types of functionality computed concurrently on GPU.
void Compute::ComputeWithCUDA(glm::vec3 *PreyPos, glm::vec3 *PreyVel, glm::vec3 *PredPos, glm::vec3 *PredVel,
                              float dt, SimulationValues *values)
{
    hipMemcpy(d_values, values, sizeof(SimulationValues), hipMemcpyHostToDevice);

    int numberOfPreyBlocks = values->Prey.Number / threadsPerBlock + 1;
    int numberOfPredatorsBlocks = values->Predators.Number / threadsPerBlock + 1;

    ComputePreyVelocities<<<numberOfPreyBlocks, threadsPerBlock>>>(PreyPos, PreyVel, d_HelperVelocitiesPrey, dt, d_values, PredPos);
    ComputePredatorsVelocities<<<numberOfPredatorsBlocks, threadsPerBlock>>>(PredPos, PredVel, d_HelperVelocitiesPredators, dt, d_values, PreyPos);
    hipDeviceSynchronize();

    SetPositions<<<numberOfPreyBlocks, threadsPerBlock>>>(PreyPos, PreyVel, d_HelperVelocitiesPrey, &d_values->Prey, dt, d_values);
    SetPositions<<<numberOfPredatorsBlocks, threadsPerBlock>>>(PredPos, PredVel, d_HelperVelocitiesPredators, &d_values->Predators, dt, d_values);
    hipDeviceSynchronize();
}

// All types of functionality computed sequentially on CPU.
void Compute::ComputeWithCPU(std::vector<glm::vec3> *pos, std::vector<glm::vec3> *vel, float dt, SimulationValues *values)
{
    for (int i = 0; i < values->Prey.Number; i++)
    {
        int count = 0;
        int smallCount = 0;
        glm::vec3 avgVel = glm::vec3(0, 0, 0);
        glm::vec3 avgPos = glm::vec3(0, 0, 0);
        glm::vec3 smallAvgPos = glm::vec3(0, 0, 0);
        for (int j = 0; j < values->Prey.Number; j++)
        {
            float distance = glm::distance((*pos)[i], (*pos)[j]);
            if (i != j && distance < values->Prey.Perception.BigPerceptionRadius)
            {
                count++;
                avgVel += (*vel)[j];
                avgPos += (*vel)[j];
                if (distance < values->Prey.Perception.BigPerceptionRadius)
                {
                    smallCount++;
                    smallAvgPos += (*pos)[j];
                }
            }
        }
        if (count > 0)
        {
            avgVel /= count;
            glm::vec3 SteeringAlignment = avgVel - (*vel)[i];
            SteeringAlignment = glm::normalize(SteeringAlignment);
            SteeringAlignment *= values->Prey.Forces.AlignmentForce;

            avgPos /= count;
            glm::vec3 SteeringCohesion = avgPos - (*pos)[i];
            SteeringCohesion = glm::normalize(SteeringCohesion);
            SteeringCohesion *= values->Prey.Forces.CohesionForce;

            (*vel)[i] += SteeringCohesion * dt;
            (*vel)[i] += SteeringAlignment * dt;
        }
        if (smallCount > 0)
        {
            smallAvgPos /= smallCount;
            glm::vec3 SteeringSeparation = (*pos)[i] - smallAvgPos;
            SteeringSeparation = glm::normalize(SteeringSeparation);
            SteeringSeparation *= values->Prey.Forces.SeparationForce;
            (*vel)[i] += SteeringSeparation * dt;
        }

        (*vel)[i] = glm::normalize((*vel)[i]);
        (*vel)[i] *= values->Prey.Speed;

        (*pos)[i] += (*vel)[i] * dt;
        if ((*pos)[i].x < -values->Environment.SizeOfContainer / 2)
            (*pos)[i].x = values->Environment.SizeOfContainer / 2;
        if ((*pos)[i].x > values->Environment.SizeOfContainer / 2)
            (*pos)[i].x = -values->Environment.SizeOfContainer / 2;
        if ((*pos)[i].y < -values->Environment.SizeOfContainer / 2)
            (*pos)[i].y = values->Environment.SizeOfContainer / 2;
        if ((*pos)[i].y > values->Environment.SizeOfContainer / 2)
            (*pos)[i].y = -values->Environment.SizeOfContainer / 2;
        if ((*pos)[i].z < -values->Environment.SizeOfContainer / 4)
            (*pos)[i].z = values->Environment.SizeOfContainer / 4;
        if ((*pos)[i].z > values->Environment.SizeOfContainer / 4)
            (*pos)[i].z = -values->Environment.SizeOfContainer / 4;
    }
}